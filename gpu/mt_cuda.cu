#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "mt_cpu.h"

#include "Timer.h"

namespace boom {

#define N_d 12


#define viscPF       (4.3e-7f*2f)	//(1.1e-6  )		// (kcal/mol)*(s/nm^2)
#define viscPF_teta  (2.3e-6f*2f)	//(1.7e-5  )		// (kcal/mol)*s

#define B_Koeff 174.0f //kcal/mol   #define B_Koeff 91.25  //kcal/mol

#define dt          (2e-10f) // s

#define dt_viscPF_teta		4.34782596e-5f				//			dt_viscPF = dt/viscPF;
#define dt_viscPF			0.000232558144f				//			dt_viscPF_teta = dt/viscPF_teta;



#define R_MT 8.128f


#define A_Koeff 53.0f
#define b_lat   12.9f

#define A_long_D 90.0f
#define b_long_D 7.9f

#define A_long_T 90.0f
#define b_long_T 7.9f

#define ro0       0.12f
#define ro0_long  0.12f

#define inv_ro0_long  8.3333f	// 1/ro0_long

#define c_lat  0.10f
#define d_lat  0.25f

#define C_Koeff 300.0f
#define Rad       2.0f

#define inv_ro0 			8.3333f 	// = 1 / ro0;
#define clat_dlat_ro0		6.6666f		// 		//clat_dlat_ro0   = 2*c_lat / (d_lat* ro0);

#define clong_dlong_ro0		6.6666f				//clong_dlong_ro0 = 2*c_lat / (d_lat* ro0);

#define 	d_lat_ro0			33.3333f		// 			d_lat_ro0 = 1 / (d_lat*ro0);


#define		d_long_ro0			33.3333f		//			d_long_ro0 = 1 / (d_lat*ro0_long);

#define pi 3.141592653f


#define fi_r 		1.3291395f
#define psi_r 		1.801785f

#define fi_l  		1.0856f
#define psi_l  		-1.339725f
#define rad_mon 	 2.0f

#define teta0_D 0.2f //rad
#define teta0_T 0.0f

#define bit char

__constant__  float Ax_1[13] = {-0.165214628f, 0.0561592989f, 0.264667839f, 0.412544012f, 0.465911359f, 0.412544012f, 0.264667839f, 0.0561594106f, -0.165214419f, -0.348739684f, -0.452372819f, -0.452372819f, -0.348739684f};

__constant__  float Ax_2[13] = {1.76747036f, 1.87652779f, 1.5556947f, 0.878470898f, 0.0f, -0.878470898f, -1.5556947f, -1.87652767f, -1.76747072f, -1.25350749f, -0.452380866f, 0.452380747f, 1.25350738f};


__constant__  float Ax_3[13] = {0.162366703f, -0.0551912338f, -0.26010555f, -0.405432671f, -0.45788008f, -0.405432671f, -0.26010555f, -0.0551913455f, 0.162366495f, 0.342728168f, 0.444574922f, 0.444574922f, 0.342728198f};


__constant__  float A_Bx_4[13] = {0.0f, 0.46472317f, 0.822983861f, 0.992708862f, 0.935016215f, 0.663122654f, 0.239315659f, -0.239315659f, -0.663122654f, -0.935016215f, -0.992708862f, -0.822983861f, -0.46472317f};


__constant__  float Ay_1[13] = {0.435634613f, 0.462514341f, 0.383437514f, 0.216519803f, 0.0f, -0.216519803f, -0.383437514f, -0.462514341f, -0.435634702f, -0.308956355f, -0.111499891f, 0.111499861f, 0.308956355f};


__constant__  float Ay_2[13] = {-0.428125232f, -0.454541624f, -0.376827925f, -0.212787479f, -0.0f, 0.212787479f, 0.376827925f, 0.454541624f, 0.428125322f, 0.30363065f, 0.109577879f, -0.10957785f, -0.30363062f};


__constant__  float Ay_3[13] = {-0.670314014f, 0.227851257f, 1.07381856f, 1.67378652f, 1.89031017f, 1.67378652f, 1.07381856f, 0.227851719f, -0.67031312f, -1.41491747f, -1.83538115f, -1.83538127f, -1.41491759f};


__constant__  float A_By_4[13] = {1.0f, 0.885456026f, 0.568064749f, 0.120536678f, -0.3546049f, -0.748510778f, -0.970941842f, -0.970941842f, -0.748510778f, -0.3546049f, 0.120536678f, 0.568064749f, 0.885456026f};


__constant__  float Az_1 = 0.465911359f;

__constant__  float Az_2 = -0.45788008f;


__constant__  float Bx_1[13] = {0.321971923f, -0.109443799f, -0.515787303f, -0.80396992f, -0.907972693f, -0.80396992f, -0.515787303f, -0.109444022f, 0.321971506f, 0.679627359f, 0.881588638f, 0.881588697f, 0.679627359f};


__constant__  float Bx_2[13] = {-1.61023343f, -1.70958889f, -1.41729772f, -0.800320745f, -0.0f, 0.800320745f, 1.41729772f, 1.70958877f, 1.61023378f, 1.14199352f, 0.412136346f, -0.412136227f, -1.1419934f};


__constant__  float Bx_3[13] = {-0.16242376f, 0.0552106313f, 0.260196954f, 0.405575156f, 0.458040982f, 0.405575156f, 0.260196954f, 0.0552107431f, -0.162423551f, -0.342848599f, -0.444731146f, -0.444731146f, -0.342848629f};


__constant__  float By_1[13] = {-0.848969102f, -0.901352584f, -0.747246861f, -0.421955943f, -0.0f, 0.421955943f, 0.747246861f, 0.901352525f, 0.848969221f, 0.602097273f, 0.2172921f, -0.217292041f, -0.602097213f};


__constant__  float By_2[13] = {0.428275675f, 0.454701364f, 0.376960337f, 0.212862253f, 0.0f, -0.212862253f, -0.376960337f, -0.454701334f, -0.428275764f, -0.303737342f, -0.109616384f, 0.109616362f, 0.303737313f};


__constant__  float By_3[13] = {0.610681832f, -0.207581252f, -0.978290021f, -1.52488387f, -1.7221452f, -1.52488387f, -0.978290021f, -0.207581669f, 0.610681057f, 1.28904426f, 1.67210281f, 1.67210281f, 1.28904426f};


__constant__  float Bz_1 = -0.907972693f;

__constant__  float Bz_2 = 0.458040982f;




__device__
__forceinline__
void calc_grad_c(		int i1, 		// i index правой молекулы
					int j1,			// j index правой молекулы

					int i2,			// i index левой молекулы

					bit type, 		// dimer type: 0 - 'D', 1 - 'T'
					bit pos,		// monomer position in dimer: 0 - bottom, 1 - top

					float x_1,		// правая молекула		mol1
					float y_1,
					float teta_1,

					float x_2,		// левая молекула		mol2
					float y_2,
					float teta_2,

					float x_3,		// верхняя молекула 	mol3
					float y_3,
					float teta_3,

					float *grad_lat_x_1,			// left component of mol1
					float *grad_lat_y_1,
					float *grad_lat_teta_1,

					float *grad_lat_x_2,			// right component of mol2
					float *grad_lat_y_2,
					float *grad_lat_teta_2,

					float *grad_long_x_1,			// up component of mol1
					float *grad_long_y_1,
					float *grad_long_teta_1,

					float *grad_long_x_3,			// down component of mol3
					float *grad_long_y_3,
					float *grad_long_teta_3

							)


{


	// теперь PE_left - это индекс i2, а PF_right - индекс i1

	float cos_t_A = cosf(teta_2);
	float sin_t_A = sinf(teta_2);
	float cos_t_B = cosf(teta_1);
	float sin_t_B = sinf(teta_1);


	float cos_t_1 = cos_t_B;
	float sin_t_1 = sin_t_B;

	float cos_t_3 = cosf(teta_3);
	float sin_t_3 = sinf(teta_3);

	// swap i1 <=> i2

	float Ax_left = Ax_1[i2]*cos_t_A + Ax_3[i2]*sin_t_A - Ax_2[i2] +
				(x_2 + R_MT) * A_Bx_4[i2];

	float Ay_left = Ay_1[i2]*cos_t_A + Ay_2[i2]*sin_t_A + Ay_3[i2] +
				(x_2 + R_MT) * A_By_4[i2];

	float Az_left = -Az_1*sin_t_A + Az_2*cos_t_A + y_2;


	float Bx_right = Bx_1[i1]*cos_t_B + Bx_3[i1]*sin_t_B - Bx_2[i1] +
				(x_1 + R_MT) * A_Bx_4[i1];

	float By_right = By_1[i1]*cos_t_B + By_2[i1]*sin_t_B + By_3[i1] +
				(x_1 + R_MT) * A_By_4[i1];

	float Bz_right = -Bz_1*sin_t_B + Bz_2*cos_t_B + y_1;

	float Dx = Ax_left - Bx_right;
	float Dy = Ay_left - By_right;
	float Dz = Az_left - Bz_right;


	float dist = sqrtf(( pow(Dx, 2) + pow(Dy, 2) + pow(Dz, 2) ));


	if (dist <=1e-7 ){
		dist = 1e-5;
	}

	float inv_dist = 1/dist;

	float drdAx = Dx * inv_dist;
	float drdAy = Dy * inv_dist;
	float drdAz = Dz * inv_dist;
	float drdBx = -drdAx;
	float drdBy = -drdAy;
	float drdBz = -drdAz;

	float dA_X_dteta = -sin_t_A*Ax_1[i2] + cos_t_A*Ax_3[i2];
	float dA_Y_dteta = -sin_t_A*Ay_1[i2] + cos_t_A*Ay_2[i2];
	float dA_Z_dteta = -cos_t_A*Az_1 - sin_t_A*Az_2;

	float drdx_A = drdAx*A_Bx_4[i2] + drdAy*A_By_4[i2];
	float drdy_A = drdAz;
	float drdteta_A = drdAx*dA_X_dteta + drdAy*dA_Y_dteta + drdAz*dA_Z_dteta;

	//================================================
	float dB_X_dteta = -sin_t_B*Bx_1[i1] + cos_t_B*Bx_3[i1];
	float dB_Y_dteta = -sin_t_B*By_1[i1] + cos_t_B*By_2[i1];
	float dB_Z_dteta = -cos_t_B*Bz_1 - sin_t_B*Bz_2;

	float drdx_B = drdBx*A_Bx_4[i1] + drdBy*A_By_4[i1];
	float drdy_B = drdBz;
	float drdteta_B = drdBx*dB_X_dteta + drdBy*dB_Y_dteta + drdBz*dB_Z_dteta;


	float Grad_U_tmp = (b_lat* dist *expf(-dist*inv_ro0)*(2.0f - dist*inv_ro0) +
				dist* clat_dlat_ro0 * expf( - (dist*dist) * d_lat_ro0 )  ) * A_Koeff;



	if ((i1==12)&&(j1>=(N_d-3))) {

		*grad_lat_x_2 = 0.0f;
		*grad_lat_y_2 = 0.0f;
		*grad_lat_teta_2 = 0.0f;

		*grad_lat_x_1 = 0.0f;
		*grad_lat_y_1 = 0.0f;
		*grad_lat_teta_1 = 0.0f;

	} else {

		*grad_lat_x_2 = Grad_U_tmp * drdx_A;
		*grad_lat_y_2 = Grad_U_tmp * drdy_A;
		*grad_lat_teta_2 = Grad_U_tmp * drdteta_A;

		*grad_lat_x_1 = Grad_U_tmp * drdx_B;
		*grad_lat_y_1 = Grad_U_tmp * drdy_B;
		*grad_lat_teta_1 = Grad_U_tmp * drdteta_B;

	}



	//	[nd] 	- 	mol3
	//	[nd-1] 	- 	mol1


	// longitudinal gradient

	float r_long_x = (x_3 - x_1) - Rad*(sin_t_1 + sin_t_3);
	float r_long_y = (y_3 - y_1) - Rad*(cos_t_1 + cos_t_3);
	float r_long = sqrtf( r_long_x*r_long_x + r_long_y*r_long_y);

	if (r_long <=1e-15 ){
		r_long = 1e-7;
	}

	float drdx_long = - r_long_x/r_long;
	float drdy_long = - r_long_y/r_long;

	float dUdr_C;

	if (pos==0) {		// bottom monomer (interaction inside dimer)
		dUdr_C = C_Koeff*r_long;
	} else {			// top monomer (interaction with upper dimer)

		float tmp1 = r_long *  expf(-r_long*inv_ro0_long)*(2 - r_long*inv_ro0_long);
		float tmp2	= r_long * clong_dlong_ro0 * expf(-(r_long*r_long) * d_long_ro0 );

		if (type==0)	// dimer type 'D'
			dUdr_C = (tmp1*b_long_D + tmp2) * A_long_D;
		else 			// dimer type 'T'
			dUdr_C = (tmp1*b_long_T + tmp2) * A_long_T;
	}



	float Grad_tmp_x = drdx_long * dUdr_C;
	float Grad_tmp_y = drdy_long * dUdr_C;

	float GradU_C_teta_1 = -dUdr_C*( drdx_long*(-Rad*cos_t_1) + drdy_long*(Rad*sin_t_1));
	float GradU_C_teta_3 =  dUdr_C*(-drdx_long*(-Rad*cos_t_3) - drdy_long*(Rad*sin_t_3));

	float Grad_tmp;
	if (type==0)		// dimer type 'D'
		Grad_tmp = B_Koeff*(teta_3 - teta_1 - teta0_D);
	else				// dimer type 'T'
		Grad_tmp = B_Koeff*(teta_3 - teta_1 - teta0_T);

	// поменял тут знак - все заработало!
	float GradU_B_teta_1 = - Grad_tmp;
	float GradU_B_teta_3 = + Grad_tmp;


	if (j1 == (N_d-1)) {

		*grad_long_x_1 		= 0.0f;
		*grad_long_y_1 		= 0.0f;
		*grad_long_teta_1	= 0.0f;

		*grad_long_x_3 		= 0.0f;
		*grad_long_y_3 		= 0.0f;
		*grad_long_teta_3	= 0.0f;

	} else {

		*grad_long_x_1 		= Grad_tmp_x;
		*grad_long_y_1 		= Grad_tmp_y;
		*grad_long_teta_1	= GradU_C_teta_1 + GradU_B_teta_1;

		*grad_long_x_3 		= - Grad_tmp_x;
		*grad_long_y_3 		= - Grad_tmp_y;
		*grad_long_teta_3	= GradU_C_teta_3 + GradU_B_teta_3;


	}


}






__global__ void
mt_cuda_kernel1(
const int          niters,                                                                   
float*    x_inout,                          
float*    y_inout,
float*    t_inout
)
{

 int i = blockDim.x * blockIdx.x + threadIdx.x;
 int j = blockDim.y * blockIdx.y + threadIdx.y;
 int s;

if(i==0 && j==0){
printf("niters %d\n", niters);
}


__shared__ float __x[13][N_d+3];
__shared__ float __y[13][N_d+3];
__shared__ float __t[13][N_d+3];

__shared__ float4 __lat_r[13][N_d+3];
__shared__ float4 __long_d[13][N_d+1];

//__lat_r[i][j]=0;
//__long_d[i][j]=0;
__x[i][j]=x_inout[i*(N_d+3) + j];     
__y[i][j]=y_inout[i*(N_d+3) + j];     
__t[i][j]=t_inout[i*(N_d+3) + j];



 float f_x, f_y, f_t;
 bit pos = 0; 
 bit type = 0;

float x_ij, y_ij, t_ij, x_i2j2,y_i2j2,t_i2j2,x_ij_1, y_ij_1, t_ij_1;

float lat_l_x, lat_l_y, lat_l_t,
lat_r_x, lat_r_y, lat_r_t,
long_u_x, long_u_y, long_u_t,
long_d_x, long_d_y, long_d_t;


int i2 = (i==12)? 0 : (i+1);
int j2 = (i==12)? (j+3) : j;
pos =(j % 2);

__syncthreads();

//#pragma unroll 2
for (s=0;s<niters;s++){

 x_ij=  __x[i][j];     y_ij=__y[i][j];     t_ij=__t[i][j];
 x_i2j2=__x[i2][j2]; y_i2j2=__y[i2][j2]; t_i2j2=__t[i2][j2];
 x_ij_1=__x[i][j+1]; y_ij_1=__y[i][j+1], t_ij_1=__t[i][j+1];

			calc_grad_c(i, j, i2, type,  pos,

				x_ij, y_ij, t_ij, 
				x_i2j2,y_i2j2,t_i2j2,
				x_ij_1, y_ij_1, t_ij_1,


				&lat_l_x,&lat_l_y,&lat_l_t,
				&lat_r_x,&lat_r_y,&lat_r_t,
				&long_u_x, &long_u_y, &long_u_t,
				&long_d_x, &long_d_y, &long_d_t);


__lat_r[i2][j2].x=lat_r_x;  __lat_r[i2][j2].y=lat_r_y;  __lat_r[i2][j2].z=lat_r_t;
__long_d[i][j+1].x=long_d_x;__long_d[i][j+1].y=long_d_y;__long_d[i][j+1].z=long_d_t;


__syncthreads();


if (j!=0){

 f_x = lat_l_x + __lat_r[i][j].x + long_u_x + __long_d[i][j].x;
 f_y = lat_l_y + __lat_r[i][j].y + long_u_y + __long_d[i][j].y;
 f_t = lat_l_t + __lat_r[i][j].z + long_u_t + __long_d[i][j].z;

 x_ij -= dt_viscPF * f_x;
 y_ij -= dt_viscPF * f_y;
 t_ij -= dt_viscPF_teta * f_t;

__x[i][j]=x_ij;     
__y[i][j]=y_ij;     
__t[i][j]=t_ij;

} // if j


__syncthreads();




} // for s



if (j!=0){
 x_inout[i*(N_d+3) + j] = __x[i][j];
 y_inout[i*(N_d+3) + j] = __y[i][j];
 t_inout[i*(N_d+3) + j] = __t[i][j];
}



}

/*
__global__ void
mt_cuda_kernel_root(
const int          niters,                                                                   
float*    x_inout,                          
float*    y_inout,
float*    t_inout
)
{
    dim3 block_size;
    block_size.x=13;
    block_size.y=N_d;

    dim3 grid_size;
    grid_size.x = 1;
    grid_size.y = 1;


    boom::mt_cuda_kernel1<<<grid_size, block_size>>>(niters, x_inout, y_inout, t_inout);
}
*/


} // ns boom



void init_coords(float x[][N_d], float y[][N_d], float t[][N_d]);
void rnd_coords(float x[][N_d], float y[][N_d], float t[][N_d]);
void cpy_coords2(float x[][N_d], float y[][N_d], float t[][N_d],float x2[][N_d+3], float y2[][N_d+3], float t2[][N_d+3]);
bool compare2(const float refData[][N_d], const float data[][N_d+3],float* err, float* delta, const float epsilon);


float x_1[13][N_d];
float y_1[13][N_d];
float t_1[13][N_d];


float x_3[13][N_d+3];
float y_3[13][N_d+3];
float t_3[13][N_d+3];


int
main(void)
{
    hipError_t err = hipSuccess;
    CPerfCounter t_cu;

    int use_cmp=0;

        float err1=0.0f;
        float delta1=0.0f;
        float epsilon = 1e-2f;

    int niters=100000;

    size_t size = 13*(N_d+3)*sizeof(float);


/*
    for (int i = 0; i < 13; ++i)
     for (int j = 0; j < N_d; ++j)
     {
        x_3[i][j] = 7;//rand()/(float)RAND_MAX;
        y_3[i][j] = 9;//rand()/(float)RAND_MAX;
        t_3[i][j] = 13;//rand()/(float)RAND_MAX;
     }
*/


	init_coords(x_1,y_1,t_1);
	rnd_coords(x_1,y_1,t_1);
        cpy_coords2(x_1,y_1,t_1,x_3,y_3,t_3);


    // Allocate the device vector
    float *d_x = NULL;
    err = hipMalloc((void **)&d_x, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector x (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device vector
    float *d_y = NULL;
    err = hipMalloc((void **)&d_y, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector y (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device vector
    float *d_t = NULL;
    err = hipMalloc((void **)&d_t, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector t (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    t_cu.Reset();t_cu.Start();


    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_x, x_3, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector x from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_y, y_3, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector y from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipMemcpy(d_t, t_3, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector t from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    dim3 block_size;
    block_size.x=13;
    block_size.y=N_d;

    dim3 grid_size;
    grid_size.x = 1;//num_elements_x / block_size.x;
    grid_size.y = 1;//num_elements_y / block_size.y;


    printf("CUDA kernel launch with %d,%d %d,%d\n", block_size.x, block_size.y, grid_size.x,grid_size.y);


    boom::mt_cuda_kernel1<<<grid_size, block_size>>>(niters, d_x, d_y, d_t);


    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
//    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(x_3, d_x, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector x from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(y_3, d_y, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector y from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(t_3, d_t, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector t from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    t_cu.Stop();

    printf("niters %d t_cu %lf\n",niters,t_cu.GetElapsedTime());


//    printf("%f,%f,%f\n",x_3[0][1],y_3[0][1],t_3[0][1]);
if(use_cmp){
	mt_cpu(niters,1,x_1,y_1,t_1,x_1,y_1,t_1);


	if(!compare2(x_1, x_3, &err1, &delta1, epsilon) || 
	   !compare2(y_1, y_3, &err1, &delta1, epsilon) ||
	   !compare2(t_1, t_3, &err1, &delta1, epsilon))
        {
		printf("Compare cu results failed (%f,%f)\n",err1,delta1);
        }
	else printf("Test OK(%f,%f)\n",err1,delta1);

}



    // Free device global memory
    err = hipFree(d_x);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_y);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_t);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    //free(h_A);
    //free(h_B);
    //free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}



void init_coords(float x[][N_d], float y[][N_d], float t[][N_d])
{
	int i,j;


	// задание y координаты для нижней спирали
	for (i=0; i<13; i++)
		y[i][0] = 2.0f*6/13*(i+1);


	// задание y координат для остальных молекул до половины высоты трубочки
	for (j=1; j<N_d-4; j++)
		for (i=0; i<13; i++)
			y[i][j] = y[i][j-1] + 2.0f*Rad;


	// задание x и teta координат так чтобы был цилиндр до половины высоты трубочки
	for (j=0; j<N_d-5; j++)
		for (i=0; i<13; i++)  {

			x[i][j] = 0.0;
			t[i][j] = 0.0;
		}


	//
	for (i=0; i<13; i++)  {

		x[i][N_d-5] = 0.6;
		t[i][N_d-5] = 0.2;

	}


	for (j=N_d-4; j<N_d; j++)
		for (i=0; i<13; i++)  {

			x[i][j] = x[i][j-1] + 2*Rad*sinf(t[i][j-1]);
			y[i][j] = y[i][j-1] + 2*Rad*cosf(t[i][j-1]);
			t[i][j] = t[i][j-1];

		}



}

void rnd_coords(float x[][N_d], float y[][N_d], float t[][N_d])
{
	int i,j;

        srand (time(NULL));


	for (i=0; i<13; i++){
	for (j=0; j<N_d; j++){
		x[i][j]+= (rand()%100)*1e-5; 
		y[i][j]-= (rand()%100)*1e-5;
		t[i][j]+= (rand()%100)*1e-5; 
	}
	}
}


void cpy_coords2(float x[][N_d], float y[][N_d], float t[][N_d],float x2[][N_d+3], float y2[][N_d+3], float t2[][N_d+3])
{
	int i,j;


	for (i=0; i<13; i++){
	for (j=0; j<N_d; j++){
		x2[i][j]= x[i][j];
		y2[i][j]= y[i][j];
		t2[i][j]= t[i][j];
	}
	}
}


bool compare2(const float refData[][N_d], const float data[][N_d+3],
             float* err, float* delta, const float epsilon = 1e-6f)
{
    float error = 0.0f;
    float ref = 0.0f;
	for(int i = 0; i<13; i++)
		for(int j = 0; j<N_d; j++) {
        float diff = refData[i][j] - data[i][j];
        if(fabs(diff)>*delta){*delta=fabs(diff);}
        error += diff * diff;
        ref += refData[i][j] * refData[i][j];
    }
    float normRef =::sqrtf((float) ref);
    if (::fabs((float) ref) < 1e-7f)
    {
        return false;
    }
    float normError = ::sqrtf((float) error);
    error = normError / normRef;
    if(error>*err)*err=error;
    return error < epsilon;
}

